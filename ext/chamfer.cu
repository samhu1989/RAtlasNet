#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

__global__ void NmDistanceKernel(int b,int n, const int dim,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	const int batch=512;
	__shared__ float buf[batch*16];
    assert( dim <= 16 );
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*dim;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*dim+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				const float* xyz1= &(xyz[(i*n+j)*dim]) ;
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&3);
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[k*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+1)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+2)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+3)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k+=4){
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[k*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+1)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+2)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+3)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}
				for (int k=end_ka;k<end_k;k++){
                    float d = 0.0;
                    for(int di=0;di<dim;++di)
                    {
                        float dif=buf[k*dim+di]-xyz1[di];
                        d += dif*dif;
                    }
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}
// int chamfer_cuda_forward(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i,float * result2,int * result2_i, hipStream_t stream){
int chamfer_cuda_forward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor dist1, at::Tensor dist2, at::Tensor idx1, at::Tensor idx2){

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
    const auto dim = xyz1.size(2);
    if( dim != xyz2.size(2) ){
        printf("dim do not match in chamfer_cuda_forward\n");
        return 0;
    }

	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, n, dim, xyz1.data<float>(), m, xyz2.data<float>(), dist1.data<float>(), idx1.data<int>());
	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, m, dim, xyz2.data<float>(), n, xyz1.data<float>(), dist2.data<float>(), idx2.data<int>());

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd updateOutput: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;


}
__global__ void NmDistanceGradKernel(int b,int n,const int dim,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
            int j2=idx1[i*n+j];
            float g=grad_dist1[i*n+j]*2;
            for(int di=0;di<dim;++di)
			{
                float x1=xyz1[(i*n+j)*dim+di];
                float x2=xyz2[(i*m+j2)*dim+di];
                atomicAdd(&(grad_xyz1[(i*n+j)*dim+di]),g*(x1-x2));
                atomicAdd(&(grad_xyz2[(i*m+j2)*dim+di]),-(g*(x1-x2)));
            }
		}
	}
}
// int chamfer_cuda_backward(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,const float * grad_dist2,const int * idx2,float * grad_xyz1,float * grad_xyz2, hipStream_t stream){
int chamfer_cuda_backward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor gradxyz1, at::Tensor gradxyz2, at::Tensor graddist1, at::Tensor graddist2, at::Tensor idx1, at::Tensor idx2){
	// hipMemset(grad_xyz1,0,b*n*3*4);
	// hipMemset(grad_xyz2,0,b*m*3*4);
	
	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
    const auto dim = xyz1.size(2);
    if( dim != xyz2.size(2) ){
        printf("dim do not match in chamfer_cuda_forward\n");
        return 0;
    }

	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,n,dim,xyz1.data<float>(),m,xyz2.data<float>(),graddist1.data<float>(),idx1.data<int>(),gradxyz1.data<float>(),gradxyz2.data<float>());
	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,m,dim,xyz2.data<float>(),n,xyz1.data<float>(),graddist2.data<float>(),idx2.data<int>(),gradxyz2.data<float>(),gradxyz1.data<float>());
	
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd get grad: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;
	
}

__device__ inline void swapf(float & a, float & b)
{   
    float tmp = a;
    a = b;
    b = tmp;
}

__device__ inline void swap(int & a, int & b)
{
    int tmp = a;
    a = b ;
    b = tmp;
}

__global__ void KnnKernel(int b,const int n,const int dim,const float * xyz,const int k,float * result,int * result_i){
    const int size = 4096;
    __shared__ float dist[size];
    __shared__ int idx[size];
    assert( n <= size );
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
    {
        for ( int i = blockIdx.y ;  i < n  ; i += gridDim.y )
        {
            for ( int j = threadIdx.x ; j < n ; j += blockDim.x )
            {
                if( i == j ){
                    dist[j] = 0;
                    idx[j]  = j;
                    continue;
                }
                float d = 0.0;
                for ( int di = 0 ; di < dim ; ++di )
                {
                    float dif = xyz[(bi*n+i)*dim+di] - xyz[(bi*n+j)*dim+di];
                    d += dif*dif;
                }
                dist[j] = d;
                idx[j] = j;
            }
            __syncthreads();
            //odd-even sort
	    int pownum = int(log2(float(n)));
	    if ( n != pow(2, pownum) ){
            for ( int cnt = 0 ; cnt < ( n + 1 ) / 2 ; ++cnt )
            {
                for ( int j = 2*threadIdx.x + 1 ; j < n ; j += 2*blockDim.x )
                {
                    if ( dist[j] < dist[ j - 1 ] )
                    {
                        swapf(dist[j], dist[j-1]);
                        swap(idx[j], idx[j-1]);
                    }
                }
                __syncthreads();
                for ( int j = 2*threadIdx.x + 2 ; j < n ; j += 2*blockDim.x )
                {
                    if ( dist[j] < dist[ j - 1 ] )
                    {
                        swapf(dist[j], dist[j-1]);
                        swap(idx[j], idx[j-1]);
                    }
                }
                __syncthreads();
            }
	    }else{	
            //Bitonic Sort
            for (unsigned int t = 2; t <= n ; t *= 2)
            {
                // Bitonic merge:
                for (unsigned int j = t / 2; j>0; j /= 2)
                {	
			for (unsigned int tid = threadIdx.x ; tid < n ; tid += blockDim.x )
                    	{
				unsigned int ixj = tid ^ j;
                    		if (ixj > tid)
                    		{
                        		if ((tid & t) == 0)
                        		{
                            			if (dist[tid] > dist[ixj])
                            			{
                                			swapf(dist[tid], dist[ixj]);
                                			swap(idx[tid], idx[ixj]);
                            			}
                        		}
                        		else
                        		{
                            			if (dist[tid] < dist[ixj])
                            			{
                                			swapf(dist[tid], dist[ixj]);
                                			swap(idx[tid], idx[ixj]);
                            			}
                        		}
                    		}
                    		
			}
			__syncthreads();	
                }
            }
	    }
            __syncthreads();
            //copy result
            for ( int j = threadIdx.x ; j < k  ; j += blockDim.x )
            {
                result[(bi*n+i)*k+j] = dist[j+1];
                result_i[ ((bi*n+i)*k+j)*2+0 ] = bi;
                result_i[ ((bi*n+i)*k+j)*2+1 ] = idx[j+1];
            }
            
        }
    }
}

int knn_cuda(at::Tensor xyz,at::Tensor k,at::Tensor dist,at::Tensor idx)
{
    const auto bs = xyz.size(0);
	const auto n = xyz.size(1); //num_points point cloud
    const auto d = xyz.size(2);
    int k_ = k.data<int>()[0];
	KnnKernel<<<dim3(bs,16,1),512>>>(bs,n,d,xyz.data<float>(),k_,dist.data<float>(),idx.data<int>());
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd Knn: %s\n", hipGetErrorString(err));
	    return 0;
	  }
	  return 1;
}

__global__ void interpKernel(const int b, const int p,const int L,const int H,const int W,const float* z,const float* prob,int* idx,float* w,float* p)
{
    float stepy = 1.0 / float(H - 1);
    float stepx = 1.0 / float(W - 1);
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                float zx = z[((bi*p+pi)*2+0)*L+li];
                float zy = z[((bi*p+pi)*2+1)*L+li];
                if( zx < 0.0 || zy < 0.0 || zx >= 1.0 || zy >= 1.0 )
                {
                    p[(bi*p+pi)*L+Li] = 0.0;
                    for( int i = 0 ; i < 4 ; i ++)
                    {
                        idx[((bi*p+pi)*2+0)*4+i)*L+li] = -1;
                        idx[((bi*p+pi)*2+1)*4+i)*L+li] = -1;
                        w[((bi*p+pi)*4+i)*L+li] = 0.0;
                    }
                    continue;
                }
                int zxn = int(zx / stepx);
                int zyn = int(zy / stepy);
                //
                idx[((bi*p+pi)*2+0)*4+0)*L+li] = zxn;
                idx[((bi*p+pi)*2+1)*4+0)*L+li] = zyn;
                idx[((bi*p+pi)*2+0)*4+1)*L+li] = zxn;
                idx[((bi*p+pi)*2+1)*4+1)*L+li] = zyn+1;
                idx[((bi*p+pi)*2+0)*4+2)*L+li] = zxn+1;
                idx[((bi*p+pi)*2+1)*4+2)*L+li] = zyn;
                idx[((bi*p+pi)*2+0)*4+3)*L+li] = zxn+1;
                idx[((bi*p+pi)*2+1)*4+3)*L+li] = zyn+1;
                //
                float x1w = zx - zxn*stepx;
                float x2w = (zxn+1)*stepx - zx;
                float y1w = zy - zyn*stepy;
                float y2w = (zyn+1)*stepy - zy;
                //
                float w1 = y2w*x2w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+0)*L+li] = w1;
                float w2 = y1w*x2w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+1)*L+li] = w2;
                float w3 = y2w*x1w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+2)*L+li] = w3
                float w4 =  y1w*x1w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+3)*L+li] = w4;
                //
                float p1 = prob[((bi*p+pi)*H+zyn)*W+zxn];
                float p2 = prob[((bi*p+pi)*H+zyn+1)*W+zxn];
                float p3 = prob[((bi*p+pi)*H+zyn)*W+zxn+1];
                float p4 = prob[((bi*p+pi)*H+zyn+1)*W+zxn+1];
                //
                p[(bi*p+pi)*L+Li] = p1*w1+p2*w2+p3*w3+p4*w4;
            }
}

int interp_cuda_forward(at::Tensor z,at::Tensor prob,at::Tensor idx,at::Tensor w,at::Tensor p)
{
    const auto b = z.size(0);
    const auto p = z.size(1); 
    const auto L = z.size(3);
    const auto H = prob.size(3);
    const auto W = prob.size(4);
    interpKernel<<<dim3(b,25,1),512>>>(b,p,L,H,W,z.data<float>(),prob.data<float>(),idx.data<int>(),w.data<float>(),p.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in nnd Knn: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void interpGradKernel(const int b, const int p,const int L,const int H,const int W,const float* grad,const int* idx,const float* w,float* gradp)
{
    float stepy = 1.0 / float(H - 1);
    float stepx = 1.0 / float(W - 1);
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                float g = grad[((bi*p+pi)*L+li];
                for(int i = 0 ; i < 4; i++)
                {
                    float wv = w[((bi*p+pi)*4+i)*L+li];
                    const int x = idx[((bi*p+pi)*2+0)*4+i)*L+li];
                    const int y = idx[((bi*p+pi)*2+1)*4+i)*L+li];
                    if((x == -1) || (y == -1))break;
                    atomicAdd(&(prob[((bi*p+pi)*H+y)*W+x]),g*wv);
                }
            }
}

int interp_cuda_backward(at::Tensor grad,at::Tensor idx,at::Tensor w,at::Tensor gradp)
{
    const auto b = grad.size(0);
    const auto p = grad.size(1); 
    const auto L = grad.size(2);
    const auto H = gradp.size(3);
    const auto W = gradp.size(4);
    interpGradKernel<<<dim3(b,25,1),512>>>(b,p,L,H,W,grad.data<float>(),idx.data<int>(),w.data<float>(),gradp.data<float>())
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in nnd Knn: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void selectKernel(const int b, const int p,const int dim,const int L,const int N,const float* in,const bool* select,int* idx,float* out)
{
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
        {
            for ( int ni = threadIdx.x; ni < N ; ni += blockDim.x )
            {
                idx[(bi*p+pi)*N+ni] = -1;
            }
            __syncthreads();
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                if( select[(bi*p+pi)*L+Li] )
                {
                    for( int ni = 0 ; ni < N ; ni ++ )
                    {
                        int v = atomicExch(&(idx[(bi*p+pi)*N+ni]),li)
                        if( v == -1 )
                        {
                            for(int di=0;di<dim;++di)
                            {
                                out[((bi*p+pi)*dim+di)*N+ni] = in[((bi*p+pi)*dim+di)*L+li];
                            }
                            break;
                        }else{
                            atomicExch(&(idx[(bi*p+pi)*N+ni]),v);
                        }
                    }
                }
            }
        }
}

int select_cuda_forward(at::Tensor in,at::Tensor select,at::Tensor idx,at::Tensor out)
{
    const auto b = in.size(0);
    const auto p = in.size(1); 
    const auto L = in.size(-1);
    const auto d = in.dim();
    const int dim = 1;
    if(d == 3)
    {
        dim = 1;
    }else if(d == 4){
        dim = in.size(2);
    }else{
        printf("input tensor must be (B,P,C,L) or (B,P,L)");
        return 0;
    }
    const auto N = out.size(-1);
    selectKernel<<<dim3(b,25,1),512>>>(b,p,dim,L,N,in.data<float>(),select.data<bool>(),idx.data<int>(),out.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in nnd Knn: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void selectGradKernel(const int b, const int p,const int dim,const int L,const int N,const float* outgrad,const int* idx,float* ingrad)
{
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
        {
            for ( int ni = threadIdx.x; ni < N ; ni += blockDim.x )
            {
                int li = idx[(bi*p+pi)*N+ni];
                for(int di=0;di<dim;++di)
                {
                    ingrad[((bi*p+pi)*dim+di)*L+li] = outgrad[((bi*p+pi)*dim+di)*N+ni];
                }
            }
        }
}

int select_cuda_backward(at::Tensor outgrad,at::Tensor idx,at::Tensor ingrad)
{
    const auto b = outgrad.size(0);
    const auto p = outgrad.size(1); 
    const auto N = outgrad.size(-1);
    const auto d = outgrad.dim();
    const auto L = ingrad.size(-1);
    const int dim = 1;
    if(d == 3)
    {
        dim = 1;
    }else if(d == 4){
        dim = in.size(2);
    }else{
        printf("input tensor must be (B,P,C,L) or (B,P,L)");
        return 0;
    }
    const auto N = out.size(-1);
    selectGradKernel<<<dim3(b,25,1),512>>>(b,p,dim,L,N,outgrad.data<float>(),idx.data<int>(),ingrad.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in nnd Knn: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

